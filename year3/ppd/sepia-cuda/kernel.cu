#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <stdio.h>
#include <iostream>

#define THREADS_PER_BLOCK 256
#define PATH_TO_ORIGIN_IMG "img3.jpg"
#define PATH_TO_RESULT_IMG "new_img.jpg"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

using namespace std;
using namespace cv;

struct pixel {
    int r, g, b;
};

void imageToVector(const Mat* img, pixel* vec)
{
    int rows = img->rows;
    int cols = img->cols;

    // convert cv::Mat pixels to structs
    int idx = 0;
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            Vec3b px = img->at<Vec3b>(Point(j, i));
            pixel mypx;
            mypx.b = px[0];
            mypx.g = px[1];
            mypx.r = px[2];
            vec[idx++] = mypx;
        }
    }
}

Mat vectorToImage(pixel* vec, int rows, int cols)
{
    Mat result_img = Mat(rows, cols, CV_8UC3);
    int idx = 0;
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            pixel mypx = vec[idx++];
            Vec3b px = Vec3b(mypx.b, mypx.g, mypx.r);
            result_img.at<Vec3b>(Point(j, i)) = px;
        }
    }
    return result_img;
}

// __device__ => called from GPU, runs on GPU
__device__ void applySepiaToPixel(pixel* px)
{
    // Pixel = Blue Green Red
    int tr = (int)(0.393 * px->r + 0.769 * px->g + 0.189 * px->b);
    int tg = (int)(0.349 * px->r + 0.686 * px->g + 0.168 * px->b);
    int tb = (int)(0.272 * px->r + 0.534 * px->g + 0.131 * px->b);
    if (tr > 255)
        px->r = 255;
    else
        px->r = tr;
    if (tg > 255)
        px->g = 255;
    else
        px->g = tg;
    if (tb > 255)
        px->b = 255;
    else
        px->b = tb;
}

// __global__ => called from CPU, runs on GPU
__global__ void sepiaKernel(pixel* d_img, int rows, int cols)
{
    // each thread applies sepia filter on every pixel of a row
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * cols;
    int total_pixels = rows * cols;

    if (start + cols < total_pixels)
        for (int y = start; y < start + cols; y++)
            applySepiaToPixel(&d_img[y]);
}

int main()
{
    pixel* h_img;    // host image
    pixel* d_img;    // device image
    int rows, cols, size;
    Mat img;

    // Read the original image on host
    char* image_path = PATH_TO_ORIGIN_IMG;
    img = imread(image_path, IMREAD_COLOR);
    if (img.empty())
    {
        cout << "Could not read the image: " << image_path << endl;
        return 1;
    }

    // Get the image dimensions
    rows = img.rows;
    cols = img.cols;
    size = rows * cols * sizeof(pixel);

    // Allocate memory on host for image
    h_img = (pixel*)malloc(size);

    // Allocate memory on device for image
    hipMalloc((void**)&d_img, size);

    // Show the original image
    imshow("Original image", img);
    waitKey(0); // Wait for a keystroke in the window

    //Convert cv::Mat to pixel vector
    imageToVector(&img, h_img);

    // Copy input image to device
    hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice);
    
    // Launch sepia kernel on GPU (each CUDA thread filters one whole row of pixels)
    int required_blocks = (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sepiaKernel CUDA_KERNEL(required_blocks, THREADS_PER_BLOCK)(d_img, rows, cols);
    
    // Copy result back to host
    hipMemcpy(h_img, d_img, size, hipMemcpyDeviceToHost);

    // Convert pixel matrix to cv::Mat
    img = vectorToImage(h_img, rows, cols);

    // Show the filtered image
    imshow("Filtered image", img);
    waitKey(0);

    // Saving the filtered image
    imwrite(PATH_TO_RESULT_IMG, img);
    
    // Cleanup
    free(h_img);
    hipFree(d_img);

    return 0;
}